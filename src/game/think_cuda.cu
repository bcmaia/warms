#include "hip/hip_runtime.h"
#include 'think_cuda.h'


// CUDA kernel
__global__ void thinkKernel(const float* mind_factor, const float* sensorial_input,
                            const float* mind_addends, float* result, size_t mind_size) {
    size_t i = blockIdx.x;
    size_t j = threadIdx.x;

    result[i] += mind_factor[i * mind_size + j] * sensorial_input[j] + mind_addends[i];
}

vectorf32 thinkCUDA(const vectorf32& mind_factor, const vectorf32& sensorial_input, const vectorf32& mind_addends) {
    vectorf32 result(OUTPUT_SIZE);

    float* d_mind_factor;
    float* d_sensorial_input;
    float* d_mind_addends;
    float* d_result;

    hipMalloc((void**)&d_mind_factor, OUTPUT_SIZE * MIND_SIZE * sizeof(float));
    hipMalloc((void**)&d_sensorial_input, MIND_SIZE * sizeof(float));
    hipMalloc((void**)&d_mind_addends, OUTPUT_SIZE * sizeof(float));
    hipMalloc((void**)&d_result, OUTPUT_SIZE * sizeof(float));

    hipMemcpy(d_mind_factor, mind_factor.data(), OUTPUT_SIZE * MIND_SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_sensorial_input, sensorial_input.data(), MIND_SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_mind_addends, mind_addends.data(), OUTPUT_SIZE * sizeof(float), hipMemcpyHostToDevice);

    dim3 gridDim(OUTPUT_SIZE);
    dim3 blockDim(MIND_SIZE);

    thinkKernel<<<gridDim, blockDim>>>(d_mind_factor, d_sensorial_input, d_mind_addends, d_result, MIND_SIZE);

    hipMemcpy(result.data(), d_result, OUTPUT_SIZE * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_mind_factor);
    hipFree(d_sensorial_input);
    hipFree(d_mind_addends);
    hipFree(d_result);

    return result;
}
